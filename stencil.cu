
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 100000
#define R 3
#define BLOCK_SIZE 1024

__global__ void singlethread_stencil(int* d_in, int* d_out, int M) {
	int tid = threadIdx.x;
	if (tid == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = -R; j <= R; j++) {
				d_out[i] += d_in[i+R+j];
			}
		}
	}
}

__global__ void multiplethreads_stencil(int* d_in, int* d_out, int M) {
	int tid = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;
	if (tid < M) {
		int result = 0;
		for (int j = -R; j <= R; j++) {
			result += d_in[tid + R + j];
		}
		d_out[tid] = result;
	}
}

__global__ void multiplethreads_stencil_perblock(int* d_in, int* d_out, int M) {
	__shared__ int temp[BLOCK_SIZE + (2 * R)];

	int g_id = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;
	int l_id = threadIdx.x + R;

	if (g_id < M) {
		temp[l_id] = d_in[g_id + R];

		if (threadIdx.x < R) {
			temp[l_id - R] = d_in[g_id];
			temp[l_id + BLOCK_SIZE] = d_in[g_id + BLOCK_SIZE];
		}

		__syncthreads();

		int result = 0;

		for (int j = -R; j <= R; j++) {
			result += temp[l_id + R];
		}
		d_out[g_id] = result;
	}
}

int main()
{
	int M = N-2*R;

	int h_in[N];
	int h_out[N-2*R];

	for (int i = 0; i < N; i++) {
		h_in[i] = 1;
	}
	int* d_in;
	int* d_out;

	//Part 1: Memory transfer from host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, M*sizeof(int));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	//Part 2: Execute kernel
	hipEventRecord(start);
	//singlethread_stencil<<<1, BLOCK_SIZE>>>(d_in, d_out, M);
	//multiplethreads_stencil<<<(int) ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
	multiplethreads_stencil_perblock<<<(int) ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	//Part 3: Memory transfer from device to host
	hipMemcpy(&h_out, d_out, M*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	//Part 4: Checking the result
	for (int i = 0; i < M; i++) {
		if (h_out[i] != 2*R+1) {
			printf("Incorrect result!\n");
			return -1;
		}
		//printf("%d ", h_out[i]);
	}
	printf("Correct result!\n");
	printf("Time used: %f milliseconds\n", milliseconds);
	return -1;
}
